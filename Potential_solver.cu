#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "cuda_wrapper.h"
#include "pointer_2d_matrix.h"

// Thread block size
#define BLOCK_SIZE  10  // number of threads in a direction of the block
#define M_WIDTH     100 // number of columns
#define M_HEIGHT    1 // number of rows

/*
// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
    int width;
    int height;
    float* elements;
} cu_Matrix;
*/

// Forward declaration of the matrix multiplication kernel
__global__ void GaussSiedel_solver_Kernel(const cu_Matrix, const cu_Matrix, cu_Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void Potential_solver(const cu_Matrix A, const cu_Matrix B, cu_Matrix C)
{
    // Load A and B to device memory
    cu_Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);
    cu_Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
               hipMemcpyHostToDevice);

    // Allocate C in device memory
    cu_Matrix d_C;
    d_C.width = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    // dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    // dim3 dimGrid(A.width / dimBlock.x, A.height / dimBlock.y);
    
    // Invoke kernel
    // dim3 dimGrid(32, 32);
    // dim3 dimBlock(38,26);
    
    ////////////////// Benchmarking  ///////////////
    
    // hipEvent_t start, stop; 
	// hipEventCreate(&start); 
	// hipEventCreate(&stop); 
	// hipEventRecord(start, 0); 
	 
	// /// your kernel call here 
    GaussSiedel_solver_Kernel<<<A.width/BLOCK_SIZE, BLOCK_SIZE>>>(d_A, d_B, d_C);
	 
	// hipEventRecord(stop, 0); 

	// hipEventSynchronize(stop); 
	 
	// float elapseTime; 
	// hipEventElapsedTime(&elapseTime, start, stop); 
	// cout << "Time to run the kernel: "<< elapseTime << " ms "<< endl;
	
    ////////////////// Benchmarking  ///////////////


    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void GaussSiedel_solver_Kernel(cu_Matrix A, cu_Matrix B, cu_Matrix C)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0;
    
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // for (int iter = 0 ; iter < 10000 ; iter++)
    // {
    //     if( (col>0) && (col<(A.width-1)) )
    //         Cvalue = 0.5*(A.elements[col+1]+A.elements[col-1]-B.elements[col]);

    //     if( col == 0 )
    //         Cvalue = 0.5*(A.elements[1]+A.elements[A.width-1]-B.elements[0]);
        
    //     if( col == (A.width-1) )
    //         Cvalue = 0.5*(A.elements[0]+A.elements[A.width-2]-B.elements[A.width-1]);

    //     __syncthreads();

    //     A.elements[col] = 0.8*Cvalue + (1.0-0.8)*A.elements[col];
    // }


    if( (col>0) && (col<(A.width-1)) )
    {
        for (int iter = 0 ; iter < 10000 ; iter++)
        {
            Cvalue = 0.5*(A.elements[col+1]+A.elements[col-1]-B.elements[col]);
            A.elements[col] = 0.8*Cvalue + (1.0-0.8)*A.elements[col];
            __syncthreads();
        }
    }

    if( col == 0 )
    {
        for (int iter = 0 ; iter < 10000 ; iter++)
        {
            Cvalue = 0.5*(A.elements[1]+A.elements[A.width-1]-B.elements[0]);
            A.elements[col] = 0.8*Cvalue + (1.0-0.8)*A.elements[col];
            __syncthreads();
        }
    }
    
    if( col == (A.width-1) )
    {
        for (int iter = 0 ; iter < 10000 ; iter++)
        {
            Cvalue = 0.5*(A.elements[0]+A.elements[A.width-2]-B.elements[A.width-1]);
            A.elements[col] = 0.8*Cvalue + (1.0-0.8)*A.elements[col];
            __syncthreads();
        }
    }


    C.elements[col] = Cvalue;
}

int get_grid_Potential(
	double* phi_grid,
	double* bvec){
	
    float *A, *B, *C;
    int i;
    cu_Matrix M_A, M_B, M_C; 

    A = (float*)malloc(M_WIDTH*sizeof(float));
    B = (float*)malloc(M_WIDTH*sizeof(float));
    C = (float*)malloc(M_WIDTH*sizeof(float));  

    srand((unsigned)time( NULL ));

    // initialize A[] and B[]
    for(i = 0; i < M_WIDTH; i++)
    {
        A[i] = (float)phi_grid[i];
        B[i] = (float)bvec[i];
        C[i] = 0.0;
    }
    
    M_A.width = M_WIDTH; M_A.height = M_HEIGHT;
    M_A.elements = A; 
    M_B.width = M_WIDTH; M_B.height = M_HEIGHT;
    M_B.elements = B; 
    M_C.width = M_WIDTH; M_C.height = M_HEIGHT;
    M_C.elements = C; 

        
    Potential_solver(M_A, M_B, M_C);
        
    
    for(i = 0; i < M_WIDTH; i++)
    {
        phi_grid[i] = (float)C[i];
        // cout << phi_grid[i] << endl;
    }
    // cin.get();

    free(A); free(B); free(C);
    return 0;
}

